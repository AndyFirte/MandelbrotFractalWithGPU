#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <complex>

#include <opencv2/highgui/highgui.hpp> //To use uchar


#include <thrust/complex.h>

#include <opencv2/core.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>

#include <chrono>
#include <iostream>

using namespace std::chrono;

using namespace std;
using cmplxDouble = complex<double>;
/*
#include "hip/hip_runtime.h"
#include ""
#include "math_functions.h"
#include <opencv2/imgproc/imgproc.hpp>

#include <stdio.h>

#include <iostream>
*/


hipError_t mandelbrotWithCuda
(
    int width, int height,
    double Ymin, double Ymax, double Xmin,
    int iter, int thresh,
    float degree
);

__device__ uchar MandelbrotIteration(thrust::complex<double> C, int iterations, float thresh, float degree)
{
    uchar greyLevel;
    thrust::complex<double> Zn = C;

    for (int i = 1; i <= iterations; i++)
    {
        Zn = Zn;
        Zn = pow(Zn, degree) + C;
        double magnitude_squared = norm(Zn); // squared magnitude of Zn
        if (magnitude_squared > thresh * thresh)
        {
            greyLevel = static_cast<uchar>(floor(255.0 - (255.0 * (i - 1) / iterations)));
            return greyLevel;
        }
        /*
        */
    }
    return 0;
}

__global__ void myKernel
(
    uchar* image, 
    int width, int height, 
    double deltaX, double deltaY,
    double Xmin, double Ymin,
    int iter, float thresh, float degree
)
{
	//columna
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	//fila 
	int j = blockDim.y * blockIdx.y + threadIdx.y;

	if (i < width && j < height) //estamos dentro de la imagen
	{
		int idx = (j * width + i) * 3;

        double real = Xmin + deltaX * i; //real number
        double imag = Ymin + deltaY * j; //imaginary number
        
        thrust::complex<double> z(real, imag);

        uchar grey = MandelbrotIteration(z, iter, thresh, degree);

        grey = 255 - grey;

        if (grey == 255)
            grey = 0;

        //uchar grey = i;

        //los floats se convierten en uchar de forma implicita.


        image[idx] = grey; //Blue
        image[idx + 1] = grey; //Green
        image[idx + 2] = grey; //Red
	}
}

int main()
{
    int k = 1;
    int M = floor(513 * k), N = floor(1024 * k);

    double Ymin = -1.1f, Ymax = 1.1f;
    double Xmin = -2.6; //Xmax is calculated with the ratio N/M

    int iter = 100;
    
    float thresh = 2;

    float degree = 2;



    //Test parameters
    M = 1080; M *= 0.9;
    N = M;

    Ymin =  0.00050;
    Ymax =  0.00600;
    Xmin = -1.77890;

    iter = 300; thresh = 2;



    // Add vectors in parallel.
    hipError_t cudaStatus = mandelbrotWithCuda(N, M, Ymin, Ymax, Xmin, iter, thresh, degree);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "mandelbrotWithCuda failed!");
        return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t mandelbrotWithCuda(
    int width, int height,
    double Ymin, double Ymax, double Xmin,
    int iter, int thresh,
    float degree
)
{
    double Yaux = Ymin; //This mirrors the image in the y axis
    Ymin = -Ymax;
    Ymax = -Yaux;

    int pixelSize = width * height;
    
    double ratio = (double)width / (double)height;

    double Xmax = Xmin + (Ymax - Ymin) * ratio;

    uchar* img_dev;

    double deltaX = (Xmax - Xmin) / (double)width;
    double deltaY = (Ymax - Ymin) / (double)height;

    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }

    // Allocate memory
    cudaStatus = hipMalloc(&img_dev, pixelSize * sizeof(uchar) * 3); //uchar and char weight 1 byte, so this multiplication is unnecessary
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

    /*
    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    */

    dim3 threads(16, 16); // = 256 pixels
    dim3 blocks(ceil((float)width / (float)threads.x), ceil((float)height / (float)threads.y));

    // Recording the timestamp at the start of the code
    auto beg = high_resolution_clock::now();

    // Launch a kernel on the GPU with one thread for each element.
    myKernel <<<blocks, threads>>>
    (
        img_dev, width, height, deltaX, deltaY, Xmin, Ymin, iter, thresh, degree
    );

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "myKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
    else
    {
        fprintf(stderr, "\nSUCCESS in hipGetLastError\n");
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    }
    else
    {
        fprintf(stderr, "\nSUCCESS in hipDeviceSynchronize\n\n");
    }

    // Taking a timestamp after the code is ran
    auto end = high_resolution_clock::now();

    auto duration = duration_cast<milliseconds>(end - beg);

    // Displaying the elapsed time
    std::cout << "\nElapsed Time: " << duration.count() << " miliseconds.\n\n\n";


    //openCV image. Use CV_8U if it's in grayscale
    cv::Mat frame = cv::Mat(cv::Size(width, height), CV_8UC3); 

    //Copy from GPU to CPU, over the image
    hipMemcpy(frame.data, img_dev, pixelSize * sizeof(uchar) * 3, hipMemcpyDeviceToHost);
    //hipMemcpy(frame.ptr(), img_dev, pixelSize * sizeof(uchar) * 3, hipMemcpyDeviceToHost); //alternative

    cv::Mat img_color;

    cv::applyColorMap(frame, img_color, cv::COLORMAP_HOT);

    cv::imshow("salida", img_color);

    cv::waitKey(0);

    return cudaStatus;
}
