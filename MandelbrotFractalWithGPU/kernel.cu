#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <opencv2/highgui/highgui.hpp> //To use uchar
/*
#include "hip/hip_runtime.h"
#include ""
#include "math_functions.h"
#include <opencv2/imgproc/imgproc.hpp>

#include <stdio.h>

#include <iostream>
*/

hipError_t addWithCuda(int width, int height);

__global__ void myKernel(int width, int height, uchar* image)
{
	//columna
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	//fila 
	int j = blockDim.y * blockIdx.y + threadIdx.y;

	if (i < width && j < height) //estamos dentro de la imagen
	{
		int idx = (j * width + i) * 3;

        //los floats se convierten en uchar de forma implicita.
        image[idx] = 255; //Blue
        image[idx + 1] = 0; //Green
        image[idx + 2] = 128; //Red
	}
}

int main()
{
    int width = 1000, height = 500;
    int pixelSize = width * height;

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(width, height);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int width, int height)
{
    int pixelSize = width * height;

    uchar* img_dev;

    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }

    // Allocate memory
    cudaStatus = hipMalloc(&img_dev, pixelSize * sizeof(uchar) * 3); //uchar and char weight 1 byte, so this multiplication is unnecessary
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

    /*
    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    */

    dim3 threads(16, 16); // = 256 pixels
    dim3 blocks(ceil((float)width / (float)threads.x), ceil((float)height / (float)threads.y));

    // Launch a kernel on the GPU with one thread for each element.
    myKernel <<<blocks, threads>>>(width, height, img_dev);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "myKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
    else
    {
        fprintf(stderr, "\nSUCCESS in hipGetLastError\n");
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    }
    else
    {
        fprintf(stderr, "\nSUCCESS in hipDeviceSynchronize\n\n");
    }

    //openCV image. Use CV_8U if it's in grayscale
    cv::Mat frame = cv::Mat(cv::Size(width, height), CV_8UC3); 

    //Copy from GPU to CPU, over the image
    hipMemcpy(frame.data, img_dev, pixelSize * sizeof(uchar) * 3, hipMemcpyDeviceToHost);
    //hipMemcpy(frame.ptr(), img_dev, pixelSize * sizeof(uchar) * 3, hipMemcpyDeviceToHost); //alternative

    cv::imshow("salida", frame);

    cv::waitKey(0);

    return cudaStatus;
}
