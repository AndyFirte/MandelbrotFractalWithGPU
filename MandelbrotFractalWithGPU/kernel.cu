#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <complex>

#include <opencv2/highgui/highgui.hpp> //To use uchar


#include <thrust/complex.h>

#include <opencv2/core.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>

#include <chrono>
#include <iostream>



#include <cmath> // for pow function

#include <iomanip> // For std::setprecision

#include "Remappers.h"

using namespace std::chrono;

using namespace std;
using cmplxDouble = complex<double>;
/*
#include "hip/hip_runtime.h"
#include ""
#include "math_functions.h"
#include <opencv2/imgproc/imgproc.hpp>

#include <stdio.h>

#include <iostream>
*/


hipError_t mandelbrotWithCuda
(
    int width, int height,
    double Ymin, double Ymax, double Xmin, double Xmax,
    int iter, int thresh,
    float degree,
    cv::Mat* fractal_image,
    bool isVideo
);


__device__ uchar MandelbrotIteration(thrust::complex<double> C, int iterations, float thresh, float degree)
{
    uchar greyLevel;
    thrust::complex<double> Zn = C;

    for (int i = 1; i <= iterations; i++)
    {
        Zn = Zn;
        Zn = pow(Zn, degree) + C;
        double magnitude_squared = norm(Zn); // squared magnitude of Zn
        if (magnitude_squared > thresh * thresh)
        {
            greyLevel = static_cast<uchar>(floor(255.0 - (255.0 * (i - 1) / iterations)));
            return greyLevel;
        }
    }
    return 0;
}

__global__ void myKernel
(
    uchar* image,
    int width, int height,
    double deltaX, double deltaY,
    double Xmin, double Ymin,
    int iter, float thresh, float degree
)
{
    //columna
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    //fila 
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    if (i < width && j < height) //estamos dentro de la imagen
    {
        int idx = (j * width + i) * 3;

        double real = Xmin + deltaX * i; //real number
        double imag = Ymin + deltaY * j; //imaginary number

        thrust::complex<double> z(real, imag);

        uchar grey = MandelbrotIteration(z, iter, thresh, degree);

        grey = 255 - grey;

        if (grey == 255)
            grey = 0;

        //uchar grey = i;

        //los floats se convierten en uchar de forma implicita.


        image[idx] = grey; //Blue
        image[idx + 1] = grey; //Green
        image[idx + 2] = grey; //Red
    }
}

int main()
{
    float k = 0.9f;
    int M = floor(1080 * k), N = floor(1080 * k);
    double ratio = (double)N / (double)M;

    double Ymin = -1.7, Ymax = 1.7;
    double Xmin = -2.4;
    double Xmax = Xmin + (Ymax - Ymin) * ratio;

    int iter = 100;

    float thresh = 2;

    float degree = 2;

    bool isVideo;
    bool repeatMainMenu = true;

    int choice;
    while (repeatMainMenu)
    {
        cout << "MENU:\n";
        cout << "1: Render image\n";
        cout << "2: Render video\n";
        cout << "Select an option: ";

        cin >> choice;

        switch (choice)
        {
        case 1:
        {
            isVideo = false;
            repeatMainMenu = false;
            break;
        }
        case 2:
        {
            isVideo = true;
            repeatMainMenu = false;
            break;
        }
        default:
        {
            cout << "Not a valid option. Try again.\n\n";
            break;
        }
        }
    }

    /*
    //Test parameters
    M = 1080; M *= 0.9;
    N = M;


    Ymin =  0.00537518315;
    Ymax =  0.00537518415;
    Xmin = -1.7763135790;

    ratio = (double)N / (double)M;

    Xmax = Xmin + (Ymax - Ymin) * ratio;

    iter = 35000; thresh = 2;

    PARAMETERS:

    Size:
            Width: 972
            Height: 972

    Y coordinates:
            Y min: 0.003582497250219543
            Y max: 0.003582498269241166

    X coordinates:
            X min: -1.477394606418177
            X max: -1.477394605399155

    Other parameters:
            Iterations: 32500
            Threshold: 2
            Degree: 2
    */


    cv::Mat fractal_image;

    int FrameStart = 0, FrameEnd = 1680; //56 seconds at 30fps
    double YminStart = Ymin, YminEnd = 0.00537518315;
    double YmaxStart = Ymax, YmaxEnd = 0.00537518415;
    double Y_limit = (YminEnd + YmaxEnd) / 2;

    double XminStart = Xmin, XminEnd = -1.7763135790;
    double XmaxEnd = XminEnd + (YmaxEnd - YminEnd) * ratio;
    double X_limit = (XminEnd + XmaxEnd) / 2;

    int iterStart = iter, iterEnd = 35000;
    float threshStart = thresh, threshEnd = 2;
    float degreeStart = degree, degreeEnd = 2;


    hipError_t cudaStatus;

    if (!isVideo)
    {
        bool exploringFractal = true;

        float zoomFactor = 6.0f;
        float shiftFactor = 6.0f;

        while (exploringFractal)
        {
            // Recording the timestamp at the start of the code
            auto beg = high_resolution_clock::now();

            cudaStatus = mandelbrotWithCuda(N, M, Ymin, Ymax, Xmin, Xmax, iter, thresh, degree, &fractal_image, isVideo);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "mandelbrotWithCuda failed!");
                return 1;
            }
            cv::imshow("salida", fractal_image);

            // Taking a timestamp after the code is ran
            auto end = high_resolution_clock::now();

            auto duration = duration_cast<milliseconds>(end - beg);

            system("cls");

            // Displaying the elapsed time
            cout << "\nElapsed Time: " << duration.count() << " miliseconds.\n";

            cout << "\nPARAMETERS:\n";

            cout << "\nSize:\n";
            cout << "\tWidth: " << N << endl;
            cout << "\tHeight: " << M << endl;
            cout << "\nY coordinates:\n";
            cout << "\tY min: " << setprecision(numeric_limits<double>::digits10 + 1) << Ymin << endl;
            cout << "\tY max: " << setprecision(numeric_limits<double>::digits10 + 1) << Ymax << endl;
            cout << "\nX coordinates:\n";
            cout << "\tX min: " << setprecision(numeric_limits<double>::digits10 + 1) << Xmin << endl;
            cout << "\tX max: " << setprecision(numeric_limits<double>::digits10 + 1) << Xmax << endl;
            cout << "\nOther parameters:\n";
            cout << "\tIterations: " << setprecision(numeric_limits<double>::digits10 + 1) << iter << endl;
            cout << "\tThreshold: " << setprecision(numeric_limits<double>::digits10 + 1) << thresh << endl;
            cout << "\tDegree: " << setprecision(numeric_limits<double>::digits10 + 1) << degree << endl;

            bool repeatExplorerMenu = true;
            while (repeatExplorerMenu)
            {
                cout << "\nEXPLORER MENU:" << endl;
                cout << "\n1: Zoom in" << endl;
                cout << "2: Zoom out" << endl;
                cout << "3: Modify zoom factor (current: " << zoomFactor << ")" << endl;
                cout << "\n4: Shift up" << endl;
                cout << "5: Shift down" << endl;
                cout << "6: Shift left" << endl;
                cout << "7: Shift right" << endl;
                cout << "8: Modify shift factor (current: " << shiftFactor << ")" << endl;
                cout << "\n9: Increase/decrease degree" << endl;
                cout << "10: Increase/decrease threshold" << endl;
                cout << "11: Increase/decrease iterations" << endl;
                cout << "12: Change resolution" << endl;
                cout << "13: Display again" << endl;
                cout << "\n0: Exit" << endl;
                cout << "\n\tSelect an option (close image window first): ";
                cv::waitKey(0);

                cin >> choice;

                repeatExplorerMenu = false;

                switch (choice)
                {
                case 0: // Exit
                {
                    exploringFractal = false;
                    break;
                }
                case 1: // Zoom in
                {
                    double deltaY = Ymax - Ymin;
                    Ymin += deltaY / zoomFactor;
                    Ymax -= deltaY / zoomFactor;

                    double deltaX = Xmax - Xmin;
                    Xmin += deltaX / zoomFactor;
                    Xmax = Xmin + (Ymax - Ymin) * ratio;
                    break;
                }
                case 2: // Zoom out
                {
                    double deltaY = Ymax - Ymin;
                    Ymin -= deltaY / (zoomFactor - 2);
                    Ymax += deltaY / (zoomFactor - 2);

                    double deltaX = Xmax - Xmin;
                    Xmin -= deltaX / (zoomFactor - 2);
                    Xmax = Xmin + (Ymax - Ymin) * ratio;
                    break;
                }
                case 3: // Modify zoom factor
                {
                    cout << "\nChoose a zoom factor." << endl;
                    cout << "It's recommended to pick a number between 3 (strong) and 10 (weak)." << endl;
                    cout << "Current zoom factor: " << zoomFactor << "." << endl;
                    cout << "New zoom factor: ";
                    cin >> zoomFactor;
                    repeatExplorerMenu = true;
                    break;
                }
                case 4: // Shift up
                {
                    double deltaY = Ymax - Ymin;
                    Ymin += deltaY / shiftFactor;
                    Ymax += deltaY / shiftFactor;
                    break;
                }
                case 5: // Shift down
                {
                    double deltaY = Ymax - Ymin;
                    Ymin -= deltaY / shiftFactor;
                    Ymax -= deltaY / shiftFactor;
                    break;
                }
                case 6: // Shift left
                {
                    double deltaX = Xmax - Xmin;
                    Xmin -= deltaX / shiftFactor;
                    Xmax -= deltaX / shiftFactor;
                    break;
                }
                case 7: // Shift right
                {
                    double deltaX = Xmax - Xmin;
                    Xmin += deltaX / shiftFactor;
                    Xmax += deltaX / shiftFactor;
                    break;
                }
                case 8: // Modify shift factor
                {
                    cout << "\nChoose a shift factor." << endl;
                    cout << "It's recommended to pick a number between 3 (strong) and 10 (weak)." << endl;
                    cout << "Current zoom factor: " << shiftFactor << "." << endl;
                    cout << "New shift factor: ";
                    cin >> shiftFactor;
                    repeatExplorerMenu = true;
                    break;
                }
                case 9: // Increment/decrement degree
                {
                    float degreeIncrement;
                    cout << "\nChoose an increment or decrement for the degree." << endl;
                    cout << "Use positive numbers for increments, negative for decrements." << endl;
                    cout << "Current degree: " << degree << "." << endl;
                    cout << "Increment by: ";
                    cin >> degreeIncrement;
                    degree += degreeIncrement;
                    break;
                }
                case 10: // Increment/decrement threshold
                {
                    float threshIncrement;
                    cout << "\nChoose an increment or decrement for the threshold." << endl;
                    cout << "Use positive numbers for increments, negative for decrements." << endl;
                    cout << "Current threshold: " << thresh << "." << endl;
                    cout << "Increment by: ";
                    cin >> threshIncrement;
                    thresh += threshIncrement;
                    break;
                }
                case 11: // Increment/decrement threshold
                {
                    int iterIncrement;
                    cout << "\nChoose an increment or decrement for the iterations." << endl;
                    cout << "Use positive integer numbers for increments, negative for decrements." << endl;
                    cout << "Current iterations: " << iter << "." << endl;
                    cout << "Increment by: ";
                    cin >> iterIncrement;
                    iter += iterIncrement;
                    break;
                }
                case 12: // Display again
                {
                    cout << "\nIntroduce the new width (the ratio will not be affected)." << endl;
                    cout << "Current width: " << N << "." << endl;
                    cout << "New width: ";
                    cin >> N;
                    M = N;
                    break;
                }
                case 13: // Display again
                {
                    break;
                }
                default:
                {
                    repeatExplorerMenu = true;
                    cout << "Not a valid option. Try again.\n\n";
                    break;
                }
                }

            }
        }
    }
    else
    {
        cout << "\nCREATING VIDEO\n\n";
        for (int index = FrameStart; index <= FrameEnd; index++)
        {
            Ymin = exponentialRemap(index, FrameEnd, YminStart, YminEnd, Y_limit);
            Ymax = exponentialRemap(index, FrameEnd, YmaxStart, YmaxEnd, Y_limit);

            Xmin = exponentialRemap(index, FrameEnd, XminStart, XminEnd, X_limit);

            iter = exponentialRemap(index, FrameEnd, iterStart, iterEnd, 0);
            thresh = exponentialRemap(index, FrameEnd, threshStart, threshEnd, 0);

            degree = linearRemap(index, FrameStart, FrameEnd, degreeStart, degreeEnd);

            cudaStatus = mandelbrotWithCuda(N, M, Ymin, Ymax, Xmin, Xmax, iter, thresh, degree, &fractal_image, isVideo);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "mandelbrotWithCuda failed!");
                return 1;
            }

            std::ostringstream ss;
            ss << "D:/Git/MandelbrotFractalWithGPU/VideoFrames/Video3/"
                << "Frame_" << index << ".png";
            std::string filename = ss.str();
            bool result = cv::imwrite(filename, fractal_image);
            if (result)
                cout << "Image " << index << " saved..." << endl;
            else
            {
                cerr << "\nError while saving image." << endl;
                return 0;
            }
        }
    }


    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t mandelbrotWithCuda(
    int width, int height,
    double Ymin, double Ymax, double Xmin, double Xmax,
    int iter, int thresh,
    float degree,
    cv::Mat* fractal_image,
    bool isVideo
)
{
    double Yaux = Ymin; //This mirrors the image in the y axis
    Ymin = -Ymax;
    Ymax = -Yaux;

    int pixelSize = width * height;

    uchar* img_dev;

    double deltaX = (Xmax - Xmin) / (double)width;
    double deltaY = (Ymax - Ymin) / (double)height;

    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }

    // Allocate memory
    cudaStatus = hipMalloc(&img_dev, pixelSize * sizeof(uchar) * 3); //uchar and char weight 1 byte, so this multiplication is unnecessary
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

    /*
    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    */

    dim3 threads(16, 16); // = 256 pixels
    dim3 blocks(ceil((float)width / (float)threads.x), ceil((float)height / (float)threads.y));

    // Launch a kernel on the GPU with one thread for each element.
    myKernel << <blocks, threads >> >
        (
            img_dev, width, height, deltaX, deltaY, Xmin, Ymin, iter, thresh, degree
            );

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "myKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
    else
    {
        if (!isVideo)
            fprintf(stderr, "\nSUCCESS in hipGetLastError\n");
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    }
    else
    {
        if (!isVideo)
            fprintf(stderr, "\nSUCCESS in hipDeviceSynchronize\n\n");
    }


    //openCV image. Use CV_8U if it's in grayscale
    cv::Mat frame = cv::Mat(cv::Size(width, height), CV_8UC3);

    //Copy from GPU to CPU, over the image
    hipMemcpy(frame.data, img_dev, pixelSize * sizeof(uchar) * 3, hipMemcpyDeviceToHost);
    //hipMemcpy(frame.ptr(), img_dev, pixelSize * sizeof(uchar) * 3, hipMemcpyDeviceToHost); //alternative

    cv::applyColorMap(frame, *fractal_image, cv::COLORMAP_HOT);

    return cudaStatus;
}
